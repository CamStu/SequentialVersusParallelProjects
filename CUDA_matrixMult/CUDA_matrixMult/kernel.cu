#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>



// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
		y[i] = x[i] + y[i];
}

//Multiplies a,b stores result in c
__global__
void MatrixMult(float *a, float *b, float *c) {
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int col = blockIdx.y*blockDim.y + threadIdx.y;
	float val = 0;

	if (row > 10000 || col > 10000)
		return;

	for (int i = 0; i < 10000; i++) {
		val += a[row * 10000 + i] * b[i * 10000 + col];
	}
	c[row * 10000 + col] = val;
}


int main(void)
{
//	int N = 1 << 20;	//Million elements
	int width = 10000, height = 10000; //10,000x10,000
	float *x, *y, *z;

	hipEvent_t start, stop;
	float time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate Unified Memory � accessible from CPU or GPU
	// While 1D arrays, these will be accessed by converting 2D index space to linear space
	hipMallocManaged(&x, width*height * sizeof(float));	
	hipMallocManaged(&y, width*height * sizeof(float));
	hipMallocManaged(&z, width*height * sizeof(float));

	//Initialize arrays on host
	for(int i=0; i < width; i++)
		for (int j = 0; j < height; j++) {
			x[i*width + j] = 1.0f;
			y[i*width + j] = 2.0f;
		}
	//A 10,000x10,000 Product matrix could be computer with one thread per entry with 100x100 blocks with 100x100 threads each
	dim3 blockSize(100, 100);
	dim3 gridSize(100, 100);

	hipEventRecord(start, 0);
	MatrixMult <<< gridSize, blockSize >>> (x, y, z);	//Block and thread dimensions chosen to be within hardware constraints
	//Timing information
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	printf("Operation complete\n");
	printf("Elapsed time on GPU= %f ms", time);
	

	// Free memory
	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}